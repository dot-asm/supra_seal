#include "hip/hip_runtime.h"
// Copyright Supranational LLC

#include <msm/batch_addition.cuh>

template __global__
void batch_addition<bucket_t>(bucket_t::mem_t ret_[],
                              const affine_t::mem_t points_[], uint32_t npoints,
                              const uint32_t bitmap[], bool accumulate,
                              uint32_t sid);

template __global__
void batch_addition<bucket_fp2_t>(bucket_fp2_t::mem_t ret_[],
                                  const affine_fp2_t::mem_t points_[],
                                  uint32_t npoints, const uint32_t bitmap[],
                                  bool accumulate, uint32_t sid);

template<class affine_t>
struct points_c {
    mutable const affine_t* points;
    size_t size;
    size_t skip;
    const uint64_t* density_map;
    size_t total_density;

    inline const affine_t& operator[](size_t i) const { return points[i]; }
};

struct msm_l_a_b_g1_b_g2_inputs_c {
    points_c<affine_t> points_l, points_a, points_b_g1;
    points_c<affine_fp2_t> points_b_g2;
    const fr_t** input_assignments, ** aux_assignments;
    size_t input_assignment_size, aux_assignment_size;
};

struct batch_add_results {
    std::vector<point_t> l;
    std::vector<point_t> a;
    std::vector<point_t> b_g1;
    std::vector<point_fp2_t> b_g2;

    batch_add_results(size_t num_circuits) : l(num_circuits),
                                             a(num_circuits),
                                             b_g1(num_circuits),
                                             b_g2(num_circuits) { }
};

const size_t CHUNK_BITS = sizeof(uint64_t) * 8; // 64 bits

#define NUM_BATCHES 8
#define GPU_DIV (32*WARP_SZ)

class split_vectors {
public:
    std::vector<std::vector<uint64_t>> bit_vector;
    std::vector<std::vector<fr_t>>     tail_msm_scalars;
    size_t batch_size, bit_vector_size;

    split_vectors(size_t num_circuits, size_t num_points)
        :   bit_vector{num_circuits},
            tail_msm_scalars{num_circuits}
    {
        batch_size = (num_points + GPU_DIV - 1) / GPU_DIV;
        batch_size = (batch_size + NUM_BATCHES - 1) / NUM_BATCHES;
        batch_size *= GPU_DIV;

        bit_vector_size = (num_points + CHUNK_BITS - 1) / CHUNK_BITS;

        for (size_t c = 0; c < num_circuits; c++) {
            bit_vector[c].resize(bit_vector_size);
        }
    }

    void tail_msms_resize(size_t num_sig_scalars) {
        size_t num_circuits = tail_msm_scalars.size();
        for (size_t c = 0; c < num_circuits; c++) {
            tail_msm_scalars[c].resize(num_sig_scalars);
        }
    }
};

template<class bucket_t,
         class point_t,
         class bucket_h = class bucket_t::mem_t,
         class affine_t = class bucket_t::affine_t,
         class affine_h = class bucket_t::affine_t::mem_t>
void execute_batch_addition(const gpu_t& gpu,
                            size_t curcuit0, size_t num_circuits,
                            const points_c<affine_t>& points,
                            const split_vectors& split_vector,
                            point_t batch_add_res[])
{
    int sm_count = gpu.sm_count();

    uint32_t nbuckets = sm_count * BATCH_ADD_BLOCK_SIZE / WARP_SZ;

    uint32_t bit_vector_size = (2 * split_vector.bit_vector[0].size() + WARP_SZ - 1) & (0u - WARP_SZ);
    size_t batch_size = split_vector.batch_size;

    size_t d_points_size = batch_size * 2 * sizeof(affine_h);
    size_t d_buckets_size = num_circuits * nbuckets * sizeof(bucket_h);

    dev_ptr_t<byte> d_temp{d_points_size + d_buckets_size +
                           num_circuits * bit_vector_size * sizeof(uint32_t)};

    assert(batch_size == (uint32_t)batch_size);

    vec2d_t<affine_h> d_points{&d_temp[0], (uint32_t)batch_size};
    vec2d_t<bucket_h> d_buckets{&d_temp[d_points_size], nbuckets};
    vec2d_t<uint32_t> d_bit_vectors{&d_temp[d_points_size + d_buckets_size],
                                    bit_vector_size};

    uint32_t sid = 0;

    for (size_t c = 0; c < num_circuits; c++)
        gpu[sid].HtoD(d_bit_vectors[c],
                      split_vector.bit_vector[curcuit0 + c].data(),
                      split_vector.bit_vector[curcuit0 + c].size() * 2);

    size_t remaining_points = points.size - points.skip;

    for (uint32_t batch = 0; remaining_points > 0; batch++, sid ^= 1) {
        uint32_t amount = std::min(remaining_points, batch_size);
        size_t cursor = batch * batch_size;

        gpu[sid].HtoD(d_points[sid], &points[cursor + points.skip], amount);

        for (size_t c = 0; c < num_circuits; c++)
            gpu[sid].launch_coop(batch_addition<bucket_t>,
                {sm_count, BATCH_ADD_BLOCK_SIZE},
                d_buckets[c], (const affine_h*)d_points[sid], amount,
                (const uint32_t*)&d_bit_vectors[c][cursor / 32],
                batch > 0, sid);

        remaining_points -= amount;
    }
    sid ^= 1;

    vec2d_t<bucket_t> buckets{nbuckets, num_circuits};
    gpu[sid].DtoH(buckets[0], d_buckets[0], num_circuits * nbuckets);
    gpu[sid].sync();

    gpu.par_map(num_circuits, 1, [&, batch_add_res, nbuckets](size_t c) {
        batch_add_res[c] = sum_up(buckets[c], nbuckets);
    });
}
