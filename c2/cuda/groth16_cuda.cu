#include "hip/hip_runtime.h"
// Copyright Supranational LLC

#include <iostream>
#include <thread>
#include <vector>

#if defined(FEATURE_BLS12_381)
# include <ff/bls12-381-fp2.hpp>
#else
# error "only FEATURE_BLS12_381 is supported"
#endif

#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>

typedef jacobian_t<fp_t> point_t;
typedef xyzz_t<fp_t> bucket_t;
typedef bucket_t::affine_t affine_t;

typedef jacobian_t<fp2_t> point_fp2_t;
typedef xyzz_t<fp2_t> bucket_fp2_t;
typedef bucket_fp2_t::affine_t affine_fp2_t;

typedef fr_t scalar_t;

#include <msm/pippenger.cuh>
#include <msm/pippenger.hpp>

#include "groth16_ntt_h.cu"
#include "groth16_split_msm.cu"

template<class point_t, class affine_t>
static void mult(point_t& ret, const affine_t point, const scalar_t& fr,
                 size_t top = scalar_t::nbits)
{
    scalar_t::pow_t scalar;
    fr.to_scalar(scalar);

    mult(ret, point, scalar, top);
}

static thread_pool_t groth16_pool;

struct msm_results {
    std::vector<point_t> h;
    std::vector<point_t> l;
    std::vector<point_t> a;
    std::vector<point_t> b_g1;
    std::vector<point_fp2_t> b_g2;

    msm_results(size_t num_circuits) : h(num_circuits),
                                       l(num_circuits),
                                       a(num_circuits),
                                       b_g1(num_circuits),
                                       b_g2(num_circuits) {}
};

struct groth16_proof {
    point_t::affine_t a;
    point_fp2_t::affine_t b;
    point_t::affine_t c;
};

#ifndef __CUDA_ARCH__

#include "groth16_srs.cuh"

#if defined(_MSC_VER) && !defined(__clang__) && !defined(__builtin_popcountll)
#define __builtin_popcountll(x) __popcnt64(x)
#endif

extern "C"
RustError generate_groth16_proof_c(const ntt_msm_h_inputs_c& ntt_msm_h_inputs,
    const msm_l_a_b_g1_b_g2_inputs_c& msm_l_a_b_g1_b_g2_inputs, size_t num_circuits,
    const fr_t r_s[], const fr_t s_s[], groth16_proof proofs[], SRS& srs)
{
    const verifying_key* vk = &srs.get_vk();

    ntt_msm_h_inputs.points_h = srs.get_h().data();
    msm_l_a_b_g1_b_g2_inputs.points_l.points = srs.get_l().data();
    msm_l_a_b_g1_b_g2_inputs.points_a.points = srs.get_a().data();
    msm_l_a_b_g1_b_g2_inputs.points_b_g1.points = srs.get_b_g1().data();
    msm_l_a_b_g1_b_g2_inputs.points_b_g2.points = srs.get_b_g2().data();

    const points_c<affine_t>& points_l = msm_l_a_b_g1_b_g2_inputs.points_l;
    const points_c<affine_t>& points_a = msm_l_a_b_g1_b_g2_inputs.points_a;
    const points_c<affine_t>& points_b_g1 = msm_l_a_b_g1_b_g2_inputs.points_b_g1;
    const points_c<affine_fp2_t>& points_b_g2 = msm_l_a_b_g1_b_g2_inputs.points_b_g2;

    split_vectors split_vectors_l{num_circuits, points_l.size};
    split_vectors split_vectors_a{num_circuits, points_a.size - points_a.skip};
    split_vectors split_vectors_b{num_circuits, points_b_g1.size - points_b_g1.skip};

    std::vector<affine_t> tail_msm_l_bases,
                          tail_msm_a_bases,
                          tail_msm_b_g1_bases;
    std::vector<affine_fp2_t> tail_msm_b_g2_bases;

    msm_results results{num_circuits};

    semaphore_t barrier;
    std::atomic<bool> caught_exception{false};
    size_t n_gpus = std::min(ngpus(), num_circuits);

    std::thread prep_msm_thread([&, num_circuits]
    {
#if 1   // minimize reference passing
        const points_c<affine_t>& points_l = msm_l_a_b_g1_b_g2_inputs.points_l;
        const points_c<affine_t>& points_a = msm_l_a_b_g1_b_g2_inputs.points_a;
        const points_c<affine_t>& points_b_g1 = msm_l_a_b_g1_b_g2_inputs.points_b_g1;
        const points_c<affine_fp2_t>& points_b_g2 = msm_l_a_b_g1_b_g2_inputs.points_b_g2;
#endif
        const fr_t** inp_assignments = msm_l_a_b_g1_b_g2_inputs.input_assignments;
        const fr_t** aux_assignments = msm_l_a_b_g1_b_g2_inputs.aux_assignments;

        size_t inp_size = msm_l_a_b_g1_b_g2_inputs.input_assignment_size;
        size_t aux_size = msm_l_a_b_g1_b_g2_inputs.aux_assignment_size;

        // pre-processing step
        // mark all significant scalars in each aux_assignment
        groth16_pool.par_map(num_circuits, [&, aux_size](size_t c) {
            auto& l_bit_vector = split_vectors_l.bit_vector[c];
            auto& a_bit_vector = split_vectors_a.bit_vector[c];
            auto& b_bit_vector = split_vectors_b.bit_vector[c];
            auto* aux_assignment = aux_assignments[c];

            size_t a_bits_cursor = 0, b_bits_cursor = 0;
            uint64_t a_bits = 0, b_bits = 0;
            uint32_t a_bit_off = 0, b_bit_off = 0;

            for (size_t i = 0; i < aux_size; i += CHUNK_BITS) {
                uint64_t a_map = points_a.density_map[i / CHUNK_BITS];
                uint64_t b_map = points_b_g1.density_map[i / CHUNK_BITS];
                uint64_t l_bits = 0;
                uint64_t map_mask = 1;

                size_t chunk_bits = std::min(CHUNK_BITS, aux_size - i);
                for (size_t j = 0; j < chunk_bits; j++, map_mask <<= 1) {
                    const fr_t& scalar = aux_assignment[i + j];

                    bool is_one = scalar.is_one();
                    bool is_zero = scalar.is_zero();

                    if (!is_zero && !is_one)
                        l_bits |= map_mask;

                    if (a_map & map_mask) {
                        if (!is_zero && !is_one)
                            a_bits |= ((uint64_t)1 << a_bit_off);

                        if (++a_bit_off == CHUNK_BITS) {
                            a_bit_off = 0;
                            a_bit_vector[a_bits_cursor++] = a_bits;
                            a_bits = 0;
                        }
                    }

                    if (b_map & map_mask) {
                        if (!is_zero && !is_one)
                            b_bits |= ((uint64_t)1 << b_bit_off);

                        if (++b_bit_off == CHUNK_BITS) {
                            b_bit_off = 0;
                            b_bit_vector[b_bits_cursor++] = b_bits;
                            b_bits = 0;
                        }
                    }
                }

                l_bit_vector[i / CHUNK_BITS] = l_bits;
            }

            if (a_bit_off)
                a_bit_vector[a_bits_cursor] = a_bits;

            if (b_bit_off)
                b_bit_vector[b_bits_cursor] = b_bits;
        });

        if (caught_exception)
            return;

        // merge all the masks from aux_assignments and count set bits
        std::vector<uint64_t> tail_msm_l_mask(split_vectors_l.bit_vector_size);
        std::vector<uint64_t> tail_msm_a_mask(split_vectors_a.bit_vector_size);
        std::vector<uint64_t> tail_msm_b_mask(split_vectors_b.bit_vector_size);

        size_t l_counter = 0,
               a_counter = points_a.skip,
               b_counter = points_b_g1.skip;

        for (size_t i = 0; i < tail_msm_l_mask.size(); i++) {
            uint64_t mask = split_vectors_l.bit_vector[0][i];
            for (size_t c = 1; c < num_circuits; c++)
                mask |= split_vectors_l.bit_vector[c][i];
            tail_msm_l_mask[i] = mask;
            l_counter += __builtin_popcountll(mask);
        }

        for (size_t i = 0; i < tail_msm_a_mask.size(); i++) {
            uint64_t mask = split_vectors_a.bit_vector[0][i];
            for (size_t c = 1; c < num_circuits; c++)
                mask |= split_vectors_a.bit_vector[c][i];
            tail_msm_a_mask[i] = mask;
            a_counter += __builtin_popcountll(mask);
        }

        for (size_t i = 0; i < tail_msm_b_mask.size(); i++) {
            uint64_t mask = split_vectors_b.bit_vector[0][i];
            for (size_t c = 1; c < num_circuits; c++)
                mask |= split_vectors_b.bit_vector[c][i];
            tail_msm_b_mask[i] = mask;
            b_counter += __builtin_popcountll(mask);
        }

        if (caught_exception)
            return;

        split_vectors_l.tail_msms_resize(l_counter);
        split_vectors_a.tail_msms_resize(a_counter);
        split_vectors_b.tail_msms_resize(b_counter);

        tail_msm_l_bases.resize(l_counter);
        tail_msm_a_bases.resize(a_counter);
        tail_msm_b_g1_bases.resize(b_counter);
        tail_msm_b_g2_bases.resize(b_counter);

        // populate bitmaps for batch additions, bases and scalars for tail msms
        groth16_pool.par_map(num_circuits, [&, inp_size, aux_size](size_t c) {
            auto& l_bit_vector = split_vectors_l.bit_vector[c];
            auto& a_bit_vector = split_vectors_a.bit_vector[c];
            auto& b_bit_vector = split_vectors_b.bit_vector[c];
            auto& tail_msm_l_scalars = split_vectors_l.tail_msm_scalars[c];
            auto& tail_msm_a_scalars = split_vectors_a.tail_msm_scalars[c];
            auto& tail_msm_b_scalars = split_vectors_b.tail_msm_scalars[c];
            auto* aux_assignment = aux_assignments[c];
            auto* inp_assignment = inp_assignments[c];

            size_t points_a_cursor = 0, points_b_cursor = 0;
            size_t l_cursor = 0;
            size_t a_cursor = 0;
            size_t b_cursor = 0;

            for (size_t i = 0; i < inp_size; i++) {
                const fr_t& scalar = inp_assignment[i];

                if (i < points_a.skip) {
                    if (c == 0)
                        tail_msm_a_bases[a_cursor] = points_a[points_a_cursor];
                    tail_msm_a_scalars[a_cursor] = scalar;
                    a_cursor++;
                    points_a_cursor++;
                }

                if (i < points_b_g1.skip) {
                    if (c == 0) {
                        tail_msm_b_g1_bases[b_cursor] = points_b_g1[points_b_cursor];
                        tail_msm_b_g2_bases[b_cursor] = points_b_g2[points_b_cursor];
                    }
                    tail_msm_b_scalars[b_cursor] = scalar;
                    b_cursor++;
                    points_b_cursor++;
                }
            }

            if (caught_exception)
                return;

            uint64_t a_mask = tail_msm_a_mask[0];
            uint64_t b_mask = tail_msm_b_mask[0];
            uint64_t a_bits = 0, b_bits = 0;
            uint32_t a_bit_off = 0, b_bit_off = 0;
            size_t a_bits_cursor = 0, b_bits_cursor = 0;

            for (size_t i = 0; i < aux_size; i += CHUNK_BITS) {
                uint64_t a_map = points_a.density_map[i / CHUNK_BITS];
                uint64_t b_map = points_b_g1.density_map[i / CHUNK_BITS];
                uint64_t l_bits = 0;
                uint64_t l_mask = tail_msm_l_mask[i / CHUNK_BITS];
                uint64_t map_mask = 1;

                size_t chunk_bits = std::min(CHUNK_BITS, aux_size - i);
                for (size_t j = 0; j < chunk_bits; j++, map_mask <<= 1) {
                    const fr_t& scalar = aux_assignment[i + j];
                    bool is_one = scalar.is_one();
                    bool is_zero = scalar.is_zero();

                    if (is_one)
                        l_bits |= map_mask;

                    if (l_mask & map_mask) {
                        if (c == 0)
                            tail_msm_l_bases[l_cursor] = points_l[i+j];
                        tail_msm_l_scalars[l_cursor] = czero(scalar, is_one);
                        l_cursor++;
                    }

                    if (a_map & map_mask) {
                        uint64_t mask = (uint64_t)1 << a_bit_off;

                        if (a_mask & mask) {
                            if (c == 0)
                                tail_msm_a_bases[a_cursor] = points_a[points_a_cursor];
                            tail_msm_a_scalars[a_cursor] = czero(scalar, is_one);
                            a_cursor++;
                        }

                        points_a_cursor++;

                        if (is_one)
                            a_bits |= mask;

                        if (++a_bit_off == CHUNK_BITS) {
                            a_bit_off = 0;
                            a_bit_vector[a_bits_cursor++] = a_bits;
                            a_bits = 0;
                            a_mask = tail_msm_a_mask[a_bits_cursor];
                        }
                    }

                    if (b_map & map_mask) {
                        uint64_t mask = (uint64_t)1 << b_bit_off;

                        if (b_mask & mask) {
                            if (c == 0) {
                                tail_msm_b_g1_bases[b_cursor] = points_b_g1[points_b_cursor];
                                tail_msm_b_g2_bases[b_cursor] = points_b_g2[points_b_cursor];
                            }
                            tail_msm_b_scalars[b_cursor] = czero(scalar, is_one);
                            b_cursor++;
                        }

                        points_b_cursor++;

                        if (is_one)
                            b_bits |= mask;

                        if (++b_bit_off == CHUNK_BITS) {
                            b_bit_off = 0;
                            b_bit_vector[b_bits_cursor++] = b_bits;
                            b_bits = 0;
                            b_mask = tail_msm_b_mask[b_bits_cursor];
                        }
                    }
                }

                l_bit_vector[i / CHUNK_BITS] = l_bits;
            }

            if (a_bit_off)
                a_bit_vector[a_bits_cursor] = a_bits;

            if (b_bit_off)
                b_bit_vector[b_bits_cursor] = b_bits;
        });
        // end of pre-processing step

        for (size_t i = 0; i < n_gpus; i++)
            barrier.notify();

        if (caught_exception)
            return;

        // tail MSM b_g2 - on CPU
        for (size_t c = 0; c < num_circuits; c++) {
            mult_pippenger<bucket_fp2_t>(results.b_g2[c],
                tail_msm_b_g2_bases, split_vectors_b.tail_msm_scalars[c],
                true, &groth16_pool);

            if (caught_exception)
                return;
        }
    });

    batch_add_results batch_add_res{num_circuits};
    std::vector<std::thread> per_gpu;
    RustError ret{hipSuccess};

    for (size_t tid = 0; tid < n_gpus; tid++) {
        per_gpu.emplace_back(std::thread([&, tid, n_gpus](size_t num_circuits)
        {
#if 1   // minimize reference passing
            const points_c<affine_t>& points_l = msm_l_a_b_g1_b_g2_inputs.points_l;
            const points_c<affine_t>& points_a = msm_l_a_b_g1_b_g2_inputs.points_a;
            const points_c<affine_t>& points_b_g1 = msm_l_a_b_g1_b_g2_inputs.points_b_g1;
            const points_c<affine_fp2_t>& points_b_g2 = msm_l_a_b_g1_b_g2_inputs.points_b_g2;
#endif
            const gpu_t& gpu = select_gpu(tid);

            size_t rem = num_circuits % n_gpus;
            num_circuits /= n_gpus;
            num_circuits += tid < rem;
            size_t circuit0 = tid * num_circuits;
            if (tid >= rem)
                circuit0 += rem;

            try {
                {
                    size_t d_a_sz = sizeof(fr_t) << ntt_msm_h_inputs.lg_domain_size;
                    gpu_ptr_t<fr_t> d_a{(scalar_t*)gpu.Dmalloc(d_a_sz)};

                    for (size_t c = 0; c < num_circuits; c++) {
                        ntt_msm_h::execute_ntt_msm_h(gpu, d_a, ntt_msm_h_inputs,
                                                     circuit0 + c, &results.h[0]);
                        if (caught_exception)
                            return;
                    }
                }

                barrier.wait();

                if (caught_exception)
                    return;

                // batch addition L - on GPU
                execute_batch_addition<bucket_t>(gpu, circuit0, num_circuits,
                                                 points_l, split_vectors_l,
                                                 &batch_add_res.l[circuit0]);
                if (caught_exception)
                    return;

                // batch addition a - on GPU
                execute_batch_addition<bucket_t>(gpu, circuit0, num_circuits,
                                                 points_a, split_vectors_a,
                                                 &batch_add_res.a[circuit0]);
                if (caught_exception)
                    return;

                // batch addition b_g1 - on GPU
                execute_batch_addition<bucket_t>(gpu, circuit0, num_circuits,
                                                 points_b_g1, split_vectors_b,
                                                 &batch_add_res.b_g1[circuit0]);
                if (caught_exception)
                    return;

                // batch addition b_g2 - on GPU
                execute_batch_addition<bucket_fp2_t>(gpu, circuit0, num_circuits,
                                                     points_b_g2, split_vectors_b,
                                                     &batch_add_res.b_g2[circuit0]);
                if (caught_exception)
                    return;

                {
                    msm_t<bucket_t, point_t, affine_t, scalar_t> msm{nullptr,
                        tail_msm_l_bases.size()};

                    for (size_t c = circuit0; c < circuit0+num_circuits; c++) {
                        // tail MSM l - on GPU
                        msm.invoke(results.l[c], tail_msm_l_bases,
                                   split_vectors_l.tail_msm_scalars[c], true);
                        if (caught_exception)
                            return;

                        // tail MSM a - on GPU
                        msm.invoke(results.a[c], tail_msm_a_bases,
                                   split_vectors_a.tail_msm_scalars[c], true);
                        if (caught_exception)
                            return;

                        // tail MSM b_g1 - on GPU
                        msm.invoke(results.b_g1[c], tail_msm_b_g1_bases,
                                   split_vectors_b.tail_msm_scalars[c], true);
                        if (caught_exception)
                            return;
                    }
                }
            } catch (const cuda_error& e) {
                bool already = caught_exception.exchange(true);
                if (!already) {
                    for (size_t i = 1; i < n_gpus; i++)
                        barrier.notify();
#ifdef TAKE_RESPONSIBILITY_FOR_ERROR_MESSAGE
                    ret = RustError{e.code(), e.what()};
#else
                    ret = RustError{e.code()};
#endif
                }
                gpu.sync();
            }
        }, num_circuits));
    }

    prep_msm_thread.join();
    for (auto& tid : per_gpu)
        tid.join();

    if (caught_exception)
        return ret;

    for (size_t circuit = 0; circuit < num_circuits; circuit++) {
        results.l[circuit].add(batch_add_res.l[circuit]);
        results.a[circuit].add(batch_add_res.a[circuit]);
        results.b_g1[circuit].add(batch_add_res.b_g1[circuit]);
        results.b_g2[circuit].add(batch_add_res.b_g2[circuit]);

        fr_t r = r_s[circuit], s = s_s[circuit];
        fr_t rs = r * s;
        // we want the scalars to be in Montomery form when passing them to
        // "mult" routine

        point_t g_a, g_c, a_answer, b1_answer, vk_delta_g1_rs, vk_alpha_g1_s,
                vk_beta_g1_r;
        point_fp2_t g_b;

        mult(vk_delta_g1_rs, vk->delta_g1, rs);
        mult(vk_alpha_g1_s, vk->alpha_g1, s);
        mult(vk_beta_g1_r, vk->beta_g1, r);

        mult(b1_answer, results.b_g1[circuit], r);

        // A
        mult(g_a, vk->delta_g1, r);
        g_a.add(vk->alpha_g1);
        g_a.add(results.a[circuit]);

        // B
        mult(g_b, vk->delta_g2, s);
        g_b.add(vk->beta_g2);
        g_b.add(results.b_g2[circuit]);

        // C
        mult(g_c, results.a[circuit], s);
        g_c.add(b1_answer);
        g_c.add(vk_delta_g1_rs);
        g_c.add(vk_alpha_g1_s);
        g_c.add(vk_beta_g1_r);
        g_c.add(results.h[circuit]);
        g_c.add(results.l[circuit]);

        // to affine
        proofs[circuit].a = g_a;
        proofs[circuit].b = g_b;
        proofs[circuit].c = g_c;
    }

    return ret;
}

#endif
