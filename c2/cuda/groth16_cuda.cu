#include "hip/hip_runtime.h"
// Copyright Supranational LLC

#include <iostream>
#include <thread>
#include <vector>
#include <mutex>
#include <algorithm>

#if defined(FEATURE_BLS12_381)
# include <ff/bls12-381-fp2.hpp>
#else
# error "only FEATURE_BLS12_381 is supported"
#endif

#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>

typedef jacobian_t<fp_t> point_t;
typedef xyzz_t<fp_t> bucket_t;
typedef bucket_t::affine_t affine_t;

typedef jacobian_t<fp2_t> point_fp2_t;
typedef xyzz_t<fp2_t> bucket_fp2_t;
typedef bucket_fp2_t::affine_t affine_fp2_t;

typedef fr_t scalar_t;

#include <msm/pippenger.cuh>
#include <msm/pippenger.hpp>

template<class Scalar>
struct Assignment {
    // Density of queries
    const uint64_t* a_aux_density;
    size_t a_aux_bit_len;
    size_t a_aux_popcount;

    const uint64_t* b_inp_density;
    size_t b_inp_bit_len;
    size_t b_inp_popcount;

    const uint64_t* b_aux_density;
    size_t b_aux_bit_len;
    size_t b_aux_popcount;

    // Evaluations of A, B, C polynomials
    const Scalar* a;
    const Scalar* b;
    const Scalar* c;
    size_t abc_size;

    // Assignments of variables
    const Scalar* inp_assignment_data;
    size_t inp_assignment_size;

    const Scalar* aux_assignment_data;
    size_t aux_assignment_size;
};

#include "groth16_ntt_h.cu"
#include "groth16_split_msm.cu"

template<class point_t, class affine_t>
static void mult(point_t& ret, const affine_t point, const scalar_t& fr,
                 size_t top = scalar_t::nbits)
{
    scalar_t::pow_t scalar;
    fr.to_scalar(scalar);

    mult(ret, point, scalar, top);
}

static thread_pool_t groth16_pool;

struct msm_results {
    std::vector<point_t> h;
    std::vector<point_t> l;
    std::vector<point_t> a;
    std::vector<point_t> b_g1;
    std::vector<point_fp2_t> b_g2;

    msm_results(size_t num_circuits) : h(num_circuits),
                                       l(num_circuits),
                                       a(num_circuits),
                                       b_g1(num_circuits),
                                       b_g2(num_circuits) {}
};

struct groth16_proof {
    point_t::affine_t a;
    point_fp2_t::affine_t b;
    point_t::affine_t c;
};

#ifndef __CUDA_ARCH__

// Mutex to control access to the GPUs
static std::mutex gpu_mtx;

#include "groth16_srs.cuh"

#if defined(_MSC_VER) && !defined(__clang__) && !defined(__builtin_popcountll)
#define __builtin_popcountll(x) __popcnt64(x)
#endif

extern "C"
RustError generate_groth16_proof_c(const ntt_msm_h_inputs_c& ntt_msm_h_inputs,
    const msm_l_a_b_g1_b_g2_inputs_c& msm_l_a_b_g1_b_g2_inputs, size_t num_circuits,
    const fr_t r_s[], const fr_t s_s[], groth16_proof proofs[], SRS& srs)
{
    std::unique_lock<std::mutex> lock(gpu_mtx);

    const verifying_key* vk = &srs.get_vk();

    ntt_msm_h_inputs.points_h = srs.get_h();
    msm_l_a_b_g1_b_g2_inputs.points_l.points = srs.get_l();
    msm_l_a_b_g1_b_g2_inputs.points_a.points = srs.get_a();
    msm_l_a_b_g1_b_g2_inputs.points_b_g1.points = srs.get_b_g1();
    msm_l_a_b_g1_b_g2_inputs.points_b_g2.points = srs.get_b_g2();

    const points_c<affine_t>& points_l = msm_l_a_b_g1_b_g2_inputs.points_l;
    const points_c<affine_t>& points_a = msm_l_a_b_g1_b_g2_inputs.points_a;
    const points_c<affine_t>& points_b_g1 = msm_l_a_b_g1_b_g2_inputs.points_b_g1;
    const points_c<affine_fp2_t>& points_b_g2 = msm_l_a_b_g1_b_g2_inputs.points_b_g2;

    split_vectors split_vectors_l{num_circuits, points_l.size};
    split_vectors split_vectors_a{num_circuits, points_a.size - points_a.skip};
    split_vectors split_vectors_b{num_circuits, points_b_g1.size - points_b_g1.skip};

    std::vector<affine_t> tail_msm_l_bases,
                          tail_msm_a_bases,
                          tail_msm_b_g1_bases;
    std::vector<affine_fp2_t> tail_msm_b_g2_bases;

    msm_results results{num_circuits};

    semaphore_t barrier;
    std::atomic<bool> caught_exception{false};
    size_t n_gpus = std::min(ngpus(), num_circuits);

    std::thread prep_msm_thread([&, num_circuits]
    {
#if 1   // minimize reference passing
        const points_c<affine_t>& points_l = msm_l_a_b_g1_b_g2_inputs.points_l;
        const points_c<affine_t>& points_a = msm_l_a_b_g1_b_g2_inputs.points_a;
        const points_c<affine_t>& points_b_g1 = msm_l_a_b_g1_b_g2_inputs.points_b_g1;
        const points_c<affine_fp2_t>& points_b_g2 = msm_l_a_b_g1_b_g2_inputs.points_b_g2;
#endif
        const fr_t** inp_assignments = msm_l_a_b_g1_b_g2_inputs.input_assignments;
        const fr_t** aux_assignments = msm_l_a_b_g1_b_g2_inputs.aux_assignments;

        size_t inp_size = msm_l_a_b_g1_b_g2_inputs.input_assignment_size;
        size_t aux_size = msm_l_a_b_g1_b_g2_inputs.aux_assignment_size;

        // pre-processing step
        // mark all significant scalars in each aux_assignment
        groth16_pool.par_map(num_circuits, [&, aux_size](size_t c) {
            auto& l_bit_vector = split_vectors_l.bit_vector[c];
            auto& a_bit_vector = split_vectors_a.bit_vector[c];
            auto& b_bit_vector = split_vectors_b.bit_vector[c];
            auto* aux_assignment = aux_assignments[c];

            size_t a_bits_cursor = 0, b_bits_cursor = 0;
            uint64_t a_bits = 0, b_bits = 0;
            uint32_t a_bit_off = 0, b_bit_off = 0;

            for (size_t i = 0; i < aux_size; i += CHUNK_BITS) {
                uint64_t a_map = points_a.density_map[i / CHUNK_BITS];
                uint64_t b_map = points_b_g1.density_map[i / CHUNK_BITS];
                uint64_t l_bits = 0;
                uint64_t map_mask = 1;

                size_t chunk_bits = std::min(CHUNK_BITS, aux_size - i);
                for (size_t j = 0; j < chunk_bits; j++, map_mask <<= 1) {
                    const fr_t& scalar = aux_assignment[i + j];

                    bool is_one = scalar.is_one();
                    bool is_zero = scalar.is_zero();

                    if (!is_zero && !is_one)
                        l_bits |= map_mask;

                    if (a_map & map_mask) {
                        if (!is_zero && !is_one)
                            a_bits |= ((uint64_t)1 << a_bit_off);

                        if (++a_bit_off == CHUNK_BITS) {
                            a_bit_off = 0;
                            a_bit_vector[a_bits_cursor++] = a_bits;
                            a_bits = 0;
                        }
                    }

                    if (b_map & map_mask) {
                        if (!is_zero && !is_one)
                            b_bits |= ((uint64_t)1 << b_bit_off);

                        if (++b_bit_off == CHUNK_BITS) {
                            b_bit_off = 0;
                            b_bit_vector[b_bits_cursor++] = b_bits;
                            b_bits = 0;
                        }
                    }
                }

                l_bit_vector[i / CHUNK_BITS] = l_bits;
            }

            if (a_bit_off)
                a_bit_vector[a_bits_cursor] = a_bits;

            if (b_bit_off)
                b_bit_vector[b_bits_cursor] = b_bits;
        });

        if (caught_exception)
            return;

        // merge all the masks from aux_assignments and count set bits
        std::vector<mask_t> tail_msm_l_mask(split_vectors_l.bit_vector_size);
        std::vector<mask_t> tail_msm_a_mask(split_vectors_a.bit_vector_size);
        std::vector<mask_t> tail_msm_b_mask(split_vectors_b.bit_vector_size);

        size_t l_counter = 0,
               a_counter = points_a.skip,
               b_counter = points_b_g1.skip;

        for (size_t i = 0; i < tail_msm_l_mask.size(); i++) {
            uint64_t mask = split_vectors_l.bit_vector[0][i];
            for (size_t c = 1; c < num_circuits; c++)
                mask |= split_vectors_l.bit_vector[c][i];
            tail_msm_l_mask[i] = mask;
            l_counter += __builtin_popcountll(mask);
        }

        for (size_t i = 0; i < tail_msm_a_mask.size(); i++) {
            uint64_t mask = split_vectors_a.bit_vector[0][i];
            for (size_t c = 1; c < num_circuits; c++)
                mask |= split_vectors_a.bit_vector[c][i];
            tail_msm_a_mask[i] = mask;
            a_counter += __builtin_popcountll(mask);
        }

        for (size_t i = 0; i < tail_msm_b_mask.size(); i++) {
            uint64_t mask = split_vectors_b.bit_vector[0][i];
            for (size_t c = 1; c < num_circuits; c++)
                mask |= split_vectors_b.bit_vector[c][i];
            tail_msm_b_mask[i] = mask;
            b_counter += __builtin_popcountll(mask);
        }

        if (caught_exception)
            return;

        split_vectors_l.tail_msms_resize(l_counter);
        split_vectors_a.tail_msms_resize(a_counter);
        split_vectors_b.tail_msms_resize(b_counter);

        tail_msm_l_bases.resize(l_counter);
        tail_msm_a_bases.resize(a_counter);
        tail_msm_b_g1_bases.resize(b_counter);
        tail_msm_b_g2_bases.resize(b_counter);

        // populate bitmaps for batch additions, bases and scalars for tail msms
        groth16_pool.par_map(num_circuits, [&, inp_size, aux_size](size_t c) {
            auto& l_bit_vector = split_vectors_l.bit_vector[c];
            auto& a_bit_vector = split_vectors_a.bit_vector[c];
            auto& b_bit_vector = split_vectors_b.bit_vector[c];
            auto& tail_msm_l_scalars = split_vectors_l.tail_msm_scalars[c];
            auto& tail_msm_a_scalars = split_vectors_a.tail_msm_scalars[c];
            auto& tail_msm_b_scalars = split_vectors_b.tail_msm_scalars[c];
            auto* aux_assignment = aux_assignments[c];
            auto* inp_assignment = inp_assignments[c];

            size_t a_cursor = 0;
            size_t b_cursor = 0;

            for (size_t i = 0; i < inp_size; i += CHUNK_BITS) {
                size_t chunk_bits = std::min(CHUNK_BITS, inp_size - i);
                uint64_t b_map = msm_l_a_b_g1_b_g2_inputs.density_map_inp[i / CHUNK_BITS];

                for (size_t j = 0; j < chunk_bits; j++, b_map >>= 1) {
                    const fr_t& scalar = inp_assignment[i + j];

                    if (b_cursor < points_b_g1.skip) {
                        if (b_map & 1) {
                            if (c == 0) {
                                tail_msm_b_g1_bases[b_cursor] = points_b_g1[b_cursor];
                                tail_msm_b_g2_bases[b_cursor] = points_b_g2[b_cursor];
                            }
                            tail_msm_b_scalars[b_cursor] = scalar;
                            b_cursor++;
                        }
                    }

                    if (a_cursor < points_a.skip) {
                        if (c == 0)
                            tail_msm_a_bases[a_cursor] = points_a[a_cursor];
                        tail_msm_a_scalars[a_cursor] = scalar;
                        a_cursor++;
                    }
                }
            }

            if (caught_exception)
                return;

            uint64_t a_mask = tail_msm_a_mask[0];
            uint64_t b_mask = tail_msm_b_mask[0];
            uint64_t a_bits = 0, b_bits = 0;
            uint32_t a_bit_off = 0, b_bit_off = 0;
            size_t a_bits_cursor = 0, b_bits_cursor = 0;
            size_t points_a_cursor = a_cursor, points_b_cursor = b_cursor;
            size_t l_cursor = 0;

            for (size_t i = 0; i < aux_size; i += CHUNK_BITS) {
                uint64_t a_map = points_a.density_map[i / CHUNK_BITS];
                uint64_t b_map = points_b_g1.density_map[i / CHUNK_BITS];
                uint64_t l_bits = 0;
                uint64_t l_mask = tail_msm_l_mask[i / CHUNK_BITS];
                uint64_t map_mask = 1;

                size_t chunk_bits = std::min(CHUNK_BITS, aux_size - i);
                for (size_t j = 0; j < chunk_bits; j++, map_mask <<= 1) {
                    const fr_t& scalar = aux_assignment[i + j];
                    bool is_one = scalar.is_one();

                    if (is_one)
                        l_bits |= map_mask;

                    if (l_mask & map_mask) {
                        if (c == 0)
                            tail_msm_l_bases[l_cursor] = points_l[i+j];
                        tail_msm_l_scalars[l_cursor] = czero(scalar, is_one);
                        l_cursor++;
                    }

                    if (a_map & map_mask) {
                        uint64_t mask = (uint64_t)1 << a_bit_off;

                        if (a_mask & mask) {
                            if (c == 0)
                                tail_msm_a_bases[a_cursor] = points_a[points_a_cursor];
                            tail_msm_a_scalars[a_cursor] = czero(scalar, is_one);
                            a_cursor++;
                        }

                        points_a_cursor++;

                        if (is_one)
                            a_bits |= mask;

                        if (++a_bit_off == CHUNK_BITS) {
                            a_bit_off = 0;
                            a_bit_vector[a_bits_cursor++] = a_bits;
                            a_bits = 0;
                            a_mask = tail_msm_a_mask[a_bits_cursor];
                        }
                    }

                    if (b_map & map_mask) {
                        uint64_t mask = (uint64_t)1 << b_bit_off;

                        if (b_mask & mask) {
                            if (c == 0) {
                                tail_msm_b_g1_bases[b_cursor] = points_b_g1[points_b_cursor];
                                tail_msm_b_g2_bases[b_cursor] = points_b_g2[points_b_cursor];
                            }
                            tail_msm_b_scalars[b_cursor] = czero(scalar, is_one);
                            b_cursor++;
                        }

                        points_b_cursor++;

                        if (is_one)
                            b_bits |= mask;

                        if (++b_bit_off == CHUNK_BITS) {
                            b_bit_off = 0;
                            b_bit_vector[b_bits_cursor++] = b_bits;
                            b_bits = 0;
                            b_mask = tail_msm_b_mask[b_bits_cursor];
                        }
                    }
                }

                l_bit_vector[i / CHUNK_BITS] = l_bits;
            }

            if (a_bit_off)
                a_bit_vector[a_bits_cursor] = a_bits;

            if (b_bit_off)
                b_bit_vector[b_bits_cursor] = b_bits;
        });
        // end of pre-processing step

        for (size_t i = 0; i < n_gpus; i++)
            barrier.notify();

        if (caught_exception)
            return;

        // tail MSM b_g2 - on CPU
        for (size_t c = 0; c < num_circuits; c++) {
            mult_pippenger<bucket_fp2_t>(results.b_g2[c],
                tail_msm_b_g2_bases, split_vectors_b.tail_msm_scalars[c],
                true, &groth16_pool);

            if (caught_exception)
                return;
        }
    });

    batch_add_results batch_add_res{num_circuits};
    std::vector<std::thread> per_gpu;
    RustError ret{hipSuccess};

    for (size_t tid = 0; tid < n_gpus; tid++) {
        per_gpu.emplace_back(std::thread([&, tid, n_gpus](size_t num_circuits)
        {
#if 1   // minimize reference passing
            const points_c<affine_t>& points_l = msm_l_a_b_g1_b_g2_inputs.points_l;
            const points_c<affine_t>& points_a = msm_l_a_b_g1_b_g2_inputs.points_a;
            const points_c<affine_t>& points_b_g1 = msm_l_a_b_g1_b_g2_inputs.points_b_g1;
            const points_c<affine_fp2_t>& points_b_g2 = msm_l_a_b_g1_b_g2_inputs.points_b_g2;
#endif
            const gpu_t& gpu = select_gpu(tid);

            size_t rem = num_circuits % n_gpus;
            num_circuits /= n_gpus;
            num_circuits += tid < rem;
            size_t circuit0 = tid * num_circuits;
            if (tid >= rem)
                circuit0 += rem;

            try {
                {
                    size_t d_a_sz = sizeof(fr_t) << ntt_msm_h_inputs.lg_domain_size;
                    gpu_ptr_t<fr_t> d_a{(scalar_t*)gpu.Dmalloc(d_a_sz)};

                    for (size_t c = 0; c < num_circuits; c++) {
                        ntt_msm_h::execute_ntt_msm_h(gpu, d_a, ntt_msm_h_inputs,
                                                     circuit0 + c, &results.h[0]);
                        if (caught_exception)
                            return;
                    }
                }

                barrier.wait();

                if (caught_exception)
                    return;

                // batch addition L - on GPU
                execute_batch_addition<bucket_t>(gpu, circuit0, num_circuits,
                                                 &points_l[0], points_l.size,
                                                 split_vectors_l,
                                                 &batch_add_res.l[circuit0]);
                if (caught_exception)
                    return;

                // batch addition a - on GPU
                execute_batch_addition<bucket_t>(gpu, circuit0, num_circuits,
                                                 &points_a[points_a.skip],
                                                 points_a.size - points_a.skip,
                                                 split_vectors_a,
                                                 &batch_add_res.a[circuit0]);
                if (caught_exception)
                    return;

                // batch addition b_g1 - on GPU
                execute_batch_addition<bucket_t>(gpu, circuit0, num_circuits,
                                                 &points_b_g1[points_b_g1.skip],
                                                 points_b_g1.size - points_b_g1.skip,
                                                 split_vectors_b,
                                                 &batch_add_res.b_g1[circuit0]);
                if (caught_exception)
                    return;

                // batch addition b_g2 - on GPU
                execute_batch_addition<bucket_fp2_t>(gpu, circuit0, num_circuits,
                                                     &points_b_g2[points_b_g2.skip],
                                                     points_b_g2.size - points_b_g2.skip,
                                                     split_vectors_b,
                                                     &batch_add_res.b_g2[circuit0]);
                if (caught_exception)
                    return;

                {
                    msm_t<bucket_t, point_t, affine_t, scalar_t> msm{nullptr,
                        tail_msm_l_bases.size()};

                    for (size_t c = circuit0; c < circuit0+num_circuits; c++) {
                        // tail MSM l - on GPU
                        msm.invoke(results.l[c], tail_msm_l_bases,
                                   split_vectors_l.tail_msm_scalars[c], true);
                        if (caught_exception)
                            return;

                        // tail MSM a - on GPU
                        msm.invoke(results.a[c], tail_msm_a_bases,
                                   split_vectors_a.tail_msm_scalars[c], true);
                        if (caught_exception)
                            return;

                        // tail MSM b_g1 - on GPU
                        msm.invoke(results.b_g1[c], tail_msm_b_g1_bases,
                                   split_vectors_b.tail_msm_scalars[c], true);
                        if (caught_exception)
                            return;
                    }
                }
            } catch (const cuda_error& e) {
                bool already = caught_exception.exchange(true);
                if (!already) {
                    for (size_t i = 1; i < n_gpus; i++)
                        barrier.notify();
#ifdef TAKE_RESPONSIBILITY_FOR_ERROR_MESSAGE
                    ret = RustError{e.code(), e.what()};
#else
                    ret = RustError{e.code()};
#endif
                }
                gpu.sync();
            }
        }, num_circuits));
    }

    prep_msm_thread.join();
    for (auto& tid : per_gpu)
        tid.join();

    if (caught_exception)
        return ret;

    for (size_t circuit = 0; circuit < num_circuits; circuit++) {
        results.l[circuit].add(batch_add_res.l[circuit]);
        results.a[circuit].add(batch_add_res.a[circuit]);
        results.b_g1[circuit].add(batch_add_res.b_g1[circuit]);
        results.b_g2[circuit].add(batch_add_res.b_g2[circuit]);

        fr_t r = r_s[circuit], s = s_s[circuit];
        fr_t rs = r * s;
        // we want the scalars to be in Montomery form when passing them to
        // "mult" routine

        point_t g_a, g_c, a_answer, b1_answer, vk_delta_g1_rs, vk_alpha_g1_s,
                vk_beta_g1_r;
        point_fp2_t g_b;

        mult(vk_delta_g1_rs, vk->delta_g1, rs);
        mult(vk_alpha_g1_s, vk->alpha_g1, s);
        mult(vk_beta_g1_r, vk->beta_g1, r);

        mult(b1_answer, results.b_g1[circuit], r);

        // A
        mult(g_a, vk->delta_g1, r);
        g_a.add(vk->alpha_g1);
        g_a.add(results.a[circuit]);

        // B
        mult(g_b, vk->delta_g2, s);
        g_b.add(vk->beta_g2);
        g_b.add(results.b_g2[circuit]);

        // C
        mult(g_c, results.a[circuit], s);
        g_c.add(b1_answer);
        g_c.add(vk_delta_g1_rs);
        g_c.add(vk_alpha_g1_s);
        g_c.add(vk_beta_g1_r);
        g_c.add(results.h[circuit]);
        g_c.add(results.l[circuit]);

        // to affine
        proofs[circuit].a = g_a;
        proofs[circuit].b = g_b;
        proofs[circuit].c = g_c;
    }

    return ret;
}

extern "C"
RustError::by_value generate_groth16_proofs_c(const Assignment<fr_t> provers[],
                                              size_t num_circuits,
                                              const fr_t r_s[], const fr_t s_s[],
                                              groth16_proof proofs[], SRS& srs)
{
    std::unique_lock<std::mutex> lock(gpu_mtx);

    const verifying_key* vk = &srs.get_vk();

    auto points_h = srs.get_h_slice();
    auto points_l = srs.get_l_slice();
    auto points_a = srs.get_a_slice();
    auto points_b_g1 = srs.get_b_g1_slice();
    auto points_b_g2 = srs.get_b_g2_slice();

    split_vectors split_vectors_l{num_circuits, points_l.size()};
    split_vectors split_vectors_a{num_circuits, points_a.size()};
    split_vectors split_vectors_b{num_circuits, points_b_g1.size()};

    std::vector<affine_t> tail_msm_l_bases,
                          tail_msm_a_bases,
                          tail_msm_b_g1_bases;
    std::vector<affine_fp2_t> tail_msm_b_g2_bases;

    msm_results results{num_circuits};

    semaphore_t barrier;
    std::atomic<bool> caught_exception{false};
    size_t n_gpus = std::min(ngpus(), num_circuits);

    std::thread prep_msm_thread([&, num_circuits]
    {
        // pre-processing step
        // mark all significant scalars in each aux_assignment
        groth16_pool.par_map(num_circuits, [&](size_t c) {
            auto& l_bit_vector = split_vectors_l.bit_vector[c];
            auto& a_bit_vector = split_vectors_a.bit_vector[c];
            auto& b_bit_vector = split_vectors_b.bit_vector[c];
            auto* a_density_map = provers[c].a_aux_density;
            auto* b_density_map = provers[c].b_aux_density;
            auto* aux_assignment = provers[c].aux_assignment_data;
            size_t aux_size = provers[c].aux_assignment_size;

            size_t a_bits_cursor = 0, b_bits_cursor = 0;
            uint64_t a_bits = 0, b_bits = 0;
            uint32_t a_bit_off = 0, b_bit_off = 0;

            for (size_t i = 0; i < aux_size; i += CHUNK_BITS) {
                uint64_t a_map = a_density_map[i / CHUNK_BITS];
                uint64_t b_map = b_density_map[i / CHUNK_BITS];
                uint64_t l_bits = 0;
                uint64_t map_mask = 1;

                size_t chunk_bits = std::min(CHUNK_BITS, aux_size - i);
                for (size_t j = 0; j < chunk_bits; j++, map_mask <<= 1) {
                    const fr_t& scalar = aux_assignment[i + j];

                    bool is_one = scalar.is_one();
                    bool is_zero = scalar.is_zero();

                    if (!is_zero && !is_one)
                        l_bits |= map_mask;

                    if (a_map & map_mask) {
                        if (!is_zero && !is_one)
                            a_bits |= ((uint64_t)1 << a_bit_off);

                        if (++a_bit_off == CHUNK_BITS) {
                            a_bit_off = 0;
                            a_bit_vector[a_bits_cursor++] = a_bits;
                            a_bits = 0;
                        }
                    }

                    if (b_map & map_mask) {
                        if (!is_zero && !is_one)
                            b_bits |= ((uint64_t)1 << b_bit_off);

                        if (++b_bit_off == CHUNK_BITS) {
                            b_bit_off = 0;
                            b_bit_vector[b_bits_cursor++] = b_bits;
                            b_bits = 0;
                        }
                    }
                }

                l_bit_vector[i / CHUNK_BITS] = l_bits;
            }

            if (a_bit_off)
                a_bit_vector[a_bits_cursor] = a_bits;

            if (b_bit_off)
                b_bit_vector[b_bits_cursor] = b_bits;
        });

        if (caught_exception)
            return;

        // merge all the masks from aux_assignments and count set bits
        std::vector<mask_t> tail_msm_l_mask(split_vectors_l.bit_vector_size);
        std::vector<mask_t> tail_msm_a_mask(split_vectors_a.bit_vector_size);
        std::vector<mask_t> tail_msm_b_mask(split_vectors_b.bit_vector_size);

        size_t l_counter = 0,
               a_counter = provers[0].inp_assignment_size,
               b_counter = provers[0].b_inp_popcount;

        for (size_t i = 0; i < tail_msm_l_mask.size(); i++) {
            uint64_t mask = split_vectors_l.bit_vector[0][i];
            for (size_t c = 1; c < num_circuits; c++)
                mask |= split_vectors_l.bit_vector[c][i];
            tail_msm_l_mask[i] = mask;
            l_counter += __builtin_popcountll(mask);
        }

        for (size_t i = 0; i < tail_msm_a_mask.size(); i++) {
            uint64_t mask = split_vectors_a.bit_vector[0][i];
            for (size_t c = 1; c < num_circuits; c++)
                mask |= split_vectors_a.bit_vector[c][i];
            tail_msm_a_mask[i] = mask;
            a_counter += __builtin_popcountll(mask);
        }

        for (size_t i = 0; i < tail_msm_b_mask.size(); i++) {
            uint64_t mask = split_vectors_b.bit_vector[0][i];
            for (size_t c = 1; c < num_circuits; c++)
                mask |= split_vectors_b.bit_vector[c][i];
            tail_msm_b_mask[i] = mask;
            b_counter += __builtin_popcountll(mask);
        }

        if (caught_exception)
            return;

        split_vectors_l.tail_msms_resize(l_counter);
        split_vectors_a.tail_msms_resize(a_counter);
        split_vectors_b.tail_msms_resize(b_counter);

        tail_msm_l_bases.resize(l_counter);
        tail_msm_a_bases.resize(a_counter);
        tail_msm_b_g1_bases.resize(b_counter);
        tail_msm_b_g2_bases.resize(b_counter);

        // populate bitmaps for batch additions, bases and scalars for tail msms
        groth16_pool.par_map(num_circuits, [&](size_t c) {
            auto& l_bit_vector = split_vectors_l.bit_vector[c];
            auto& a_bit_vector = split_vectors_a.bit_vector[c];
            auto& b_bit_vector = split_vectors_b.bit_vector[c];
            auto& tail_msm_l_scalars = split_vectors_l.tail_msm_scalars[c];
            auto& tail_msm_a_scalars = split_vectors_a.tail_msm_scalars[c];
            auto& tail_msm_b_scalars = split_vectors_b.tail_msm_scalars[c];
            auto* aux_assignment = provers[c].aux_assignment_data;
            auto* inp_assignment = provers[c].inp_assignment_data;
            size_t aux_size = provers[c].aux_assignment_size;
            size_t inp_size = provers[c].inp_assignment_size;

            size_t a_cursor = 0;
            size_t b_cursor = 0;

            for (size_t i = 0; i < inp_size; i += CHUNK_BITS) {
                size_t chunk_bits = std::min(CHUNK_BITS, inp_size - i);
                uint64_t b_map = provers[c].b_inp_density[i / CHUNK_BITS];

                for (size_t j = 0; j < chunk_bits; j++, b_map >>= 1) {
                    const fr_t& scalar = inp_assignment[i + j];

                    if (b_cursor < provers[c].b_inp_popcount) {
                        if (b_map & 1) {
                            if (c == 0) {
                                tail_msm_b_g1_bases[b_cursor] = points_b_g1[b_cursor];
                                tail_msm_b_g2_bases[b_cursor] = points_b_g2[b_cursor];
                            }
                            tail_msm_b_scalars[b_cursor] = scalar;
                            b_cursor++;
                        }
                    }

                    if (a_cursor < provers[c].inp_assignment_size) {
                        if (c == 0)
                            tail_msm_a_bases[a_cursor] = points_a[a_cursor];
                        tail_msm_a_scalars[a_cursor] = scalar;
                        a_cursor++;
                    }
                }
            }

            if (caught_exception)
                return;

            uint64_t a_mask = tail_msm_a_mask[0];
            uint64_t b_mask = tail_msm_b_mask[0];
            uint64_t a_bits = 0, b_bits = 0;
            uint32_t a_bit_off = 0, b_bit_off = 0;
            size_t a_bits_cursor = 0, b_bits_cursor = 0;
            size_t points_a_cursor = a_cursor, points_b_cursor = b_cursor;
            size_t l_cursor = 0;
            auto* a_density_map = provers[c].a_aux_density;
            auto* b_density_map = provers[c].b_aux_density;

            for (size_t i = 0; i < aux_size; i += CHUNK_BITS) {
                uint64_t a_map = a_density_map[i / CHUNK_BITS];
                uint64_t b_map = b_density_map[i / CHUNK_BITS];
                uint64_t l_bits = 0;
                uint64_t l_mask = tail_msm_l_mask[i / CHUNK_BITS];
                uint64_t map_mask = 1;

                size_t chunk_bits = std::min(CHUNK_BITS, aux_size - i);
                for (size_t j = 0; j < chunk_bits; j++, map_mask <<= 1) {
                    const fr_t& scalar = aux_assignment[i + j];
                    bool is_one = scalar.is_one();

                    if (is_one)
                        l_bits |= map_mask;

                    if (l_mask & map_mask) {
                        if (c == 0)
                            tail_msm_l_bases[l_cursor] = points_l[i+j];
                        tail_msm_l_scalars[l_cursor] = czero(scalar, is_one);
                        l_cursor++;
                    }

                    if (a_map & map_mask) {
                        uint64_t mask = (uint64_t)1 << a_bit_off;

                        if (a_mask & mask) {
                            if (c == 0)
                                tail_msm_a_bases[a_cursor] = points_a[points_a_cursor];
                            tail_msm_a_scalars[a_cursor] = czero(scalar, is_one);
                            a_cursor++;
                        }

                        points_a_cursor++;

                        if (is_one)
                            a_bits |= mask;

                        if (++a_bit_off == CHUNK_BITS) {
                            a_bit_off = 0;
                            a_bit_vector[a_bits_cursor++] = a_bits;
                            a_bits = 0;
                            a_mask = tail_msm_a_mask[a_bits_cursor];
                        }
                    }

                    if (b_map & map_mask) {
                        uint64_t mask = (uint64_t)1 << b_bit_off;

                        if (b_mask & mask) {
                            if (c == 0) {
                                tail_msm_b_g1_bases[b_cursor] = points_b_g1[points_b_cursor];
                                tail_msm_b_g2_bases[b_cursor] = points_b_g2[points_b_cursor];
                            }
                            tail_msm_b_scalars[b_cursor] = czero(scalar, is_one);
                            b_cursor++;
                        }

                        points_b_cursor++;

                        if (is_one)
                            b_bits |= mask;

                        if (++b_bit_off == CHUNK_BITS) {
                            b_bit_off = 0;
                            b_bit_vector[b_bits_cursor++] = b_bits;
                            b_bits = 0;
                            b_mask = tail_msm_b_mask[b_bits_cursor];
                        }
                    }
                }

                l_bit_vector[i / CHUNK_BITS] = l_bits;
            }

            if (a_bit_off)
                a_bit_vector[a_bits_cursor] = a_bits;

            if (b_bit_off)
                b_bit_vector[b_bits_cursor] = b_bits;
        });
        // end of pre-processing step

        for (size_t i = 0; i < n_gpus; i++)
            barrier.notify();

        if (caught_exception)
            return;

        // tail MSM b_g2 - on CPU
        for (size_t c = 0; c < num_circuits; c++) {
            mult_pippenger<bucket_fp2_t>(results.b_g2[c],
                tail_msm_b_g2_bases, split_vectors_b.tail_msm_scalars[c],
                true, &groth16_pool);

            if (caught_exception)
                return;
        }
    });

    batch_add_results batch_add_res{num_circuits};
    std::vector<std::thread> per_gpu;
    RustError ret{hipSuccess};

    for (size_t tid = 0; tid < n_gpus; tid++) {
        per_gpu.emplace_back(std::thread([&, tid, n_gpus](size_t num_circuits)
        {
            const gpu_t& gpu = select_gpu(tid);

            size_t rem = num_circuits % n_gpus;
            num_circuits /= n_gpus;
            num_circuits += tid < rem;
            size_t circuit0 = tid * num_circuits;
            if (tid >= rem)
                circuit0 += rem;

            try {
                {
                    size_t d_a_sz = sizeof(fr_t) << (lg2(provers[0].abc_size - 1) + 1);
                    gpu_ptr_t<fr_t> d_a{(scalar_t*)gpu.Dmalloc(d_a_sz)};

                    for (size_t c = circuit0; c < circuit0 + num_circuits; c++) {
                        ntt_msm_h::execute_ntt_msm_h(gpu, d_a, provers[c],
                                                     &points_h[0], results.h[c]);
                        if (caught_exception)
                            return;
                    }
                }

                barrier.wait();

                if (caught_exception)
                    return;

                // batch addition L - on GPU
                execute_batch_addition<bucket_t>(gpu, circuit0, num_circuits,
                                                 &points_l[0],
                                                 provers[0].aux_assignment_size,
                                                 split_vectors_l,
                                                 &batch_add_res.l[circuit0]);
                if (caught_exception)
                    return;

                // batch addition a - on GPU
                execute_batch_addition<bucket_t>(gpu, circuit0, num_circuits,
                                                 &points_a[provers[0].inp_assignment_size],
                                                 provers[0].a_aux_popcount,
                                                 split_vectors_a,
                                                 &batch_add_res.a[circuit0]);
                if (caught_exception)
                    return;

                // batch addition b_g1 - on GPU
                execute_batch_addition<bucket_t>(gpu, circuit0, num_circuits,
                                                 &points_b_g1[provers[0].b_inp_popcount],
                                                 provers[0].b_aux_popcount,
                                                 split_vectors_b,
                                                 &batch_add_res.b_g1[circuit0]);
                if (caught_exception)
                    return;

                // batch addition b_g2 - on GPU
                execute_batch_addition<bucket_fp2_t>(gpu, circuit0, num_circuits,
                                                     &points_b_g2[provers[0].b_inp_popcount],
                                                     provers[0].b_aux_popcount,
                                                     split_vectors_b,
                                                     &batch_add_res.b_g2[circuit0]);
                if (caught_exception)
                    return;

                {
                    msm_t<bucket_t, point_t, affine_t, scalar_t> msm{nullptr,
                        tail_msm_l_bases.size()};

                    for (size_t c = circuit0; c < circuit0+num_circuits; c++) {
                        // tail MSM l - on GPU
                        msm.invoke(results.l[c], tail_msm_l_bases,
                                   split_vectors_l.tail_msm_scalars[c], true);
                        if (caught_exception)
                            return;

                        // tail MSM a - on GPU
                        msm.invoke(results.a[c], tail_msm_a_bases,
                                   split_vectors_a.tail_msm_scalars[c], true);
                        if (caught_exception)
                            return;

                        // tail MSM b_g1 - on GPU
                        msm.invoke(results.b_g1[c], tail_msm_b_g1_bases,
                                   split_vectors_b.tail_msm_scalars[c], true);
                        if (caught_exception)
                            return;
                    }
                }
            } catch (const cuda_error& e) {
                bool already = caught_exception.exchange(true);
                if (!already) {
                    for (size_t i = 1; i < n_gpus; i++)
                        barrier.notify();
#ifdef TAKE_RESPONSIBILITY_FOR_ERROR_MESSAGE
                    ret = RustError{e.code(), e.what()};
#else
                    ret = RustError{e.code()};
#endif
                }
                gpu.sync();
            }
        }, num_circuits));
    }

    prep_msm_thread.join();
    for (auto& tid : per_gpu)
        tid.join();

    if (caught_exception)
        return ret;

    for (size_t circuit = 0; circuit < num_circuits; circuit++) {
        results.l[circuit].add(batch_add_res.l[circuit]);
        results.a[circuit].add(batch_add_res.a[circuit]);
        results.b_g1[circuit].add(batch_add_res.b_g1[circuit]);
        results.b_g2[circuit].add(batch_add_res.b_g2[circuit]);

        fr_t r = r_s[circuit], s = s_s[circuit];
        fr_t rs = r * s;
        // we want the scalars to be in Montomery form when passing them to
        // "mult" routine

        point_t g_a, g_c, a_answer, b1_answer, vk_delta_g1_rs, vk_alpha_g1_s,
                vk_beta_g1_r;
        point_fp2_t g_b;

        mult(vk_delta_g1_rs, vk->delta_g1, rs);
        mult(vk_alpha_g1_s, vk->alpha_g1, s);
        mult(vk_beta_g1_r, vk->beta_g1, r);

        mult(b1_answer, results.b_g1[circuit], r);

        // A
        mult(g_a, vk->delta_g1, r);
        g_a.add(vk->alpha_g1);
        g_a.add(results.a[circuit]);

        // B
        mult(g_b, vk->delta_g2, s);
        g_b.add(vk->beta_g2);
        g_b.add(results.b_g2[circuit]);

        // C
        mult(g_c, results.a[circuit], s);
        g_c.add(b1_answer);
        g_c.add(vk_delta_g1_rs);
        g_c.add(vk_alpha_g1_s);
        g_c.add(vk_beta_g1_r);
        g_c.add(results.h[circuit]);
        g_c.add(results.l[circuit]);

        // to affine
        proofs[circuit].a = g_a;
        proofs[circuit].b = g_b;
        proofs[circuit].c = g_c;
    }

    return ret;
}

#endif
